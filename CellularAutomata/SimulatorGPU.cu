#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "SimulatorGPU.hpp"
#include "IRulesArray.hpp"
#include "RulesArrayConway.hpp"
#include "RulesArrayBML.hpp"
#include <>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>

#define Y_DIM context[0]
#define X_DIM context[1]
#define NUM_SEGMENTS context[2]


namespace CellularAutomata {

	



	template <typename T>
	SimulatorGPU<T>::~SimulatorGPU()
	{
		checkCudaErrors(hipDeviceReset());
	}

	/** Construct an instance of RulesArrayConway on the device
		@param dest: Destination address for the ruleset. Must already be (cuda)Malloced
		@param args: 2-element array containing: [y_dim, x_dim] of the frames to be simulated
	 */
	template <typename T>
	__global__ void constructConway(RulesArrayConway<T>* dest, int* args)
	{
		printf("\nydim: %d, xdIM: %d\n", args[0], args[1]);
		new (dest) RulesArrayConway<T>(args[0],args[1]);
	}

	/** Construct an instance of RulesArrayBML on the device
		@param dest: Destination address for the ruleset. Must already be (cuda)Malloced
		@param args: 2-element array containing: [y_dim, x_dim] of the frames to be simulated
	 */
	template <typename T>
	__global__ void constructBML(RulesArrayBML<T>* dest, int* args)
	{
		printf("\nydim: %d, xdIM: %d\n", args[0], args[1]);
		new (dest) RulesArrayBML<T>(args[0], args[1]);
	}

	/** Step forward the given region, using the given ruleset
		@param A: The previous frame of the simulation
		@param B: The frame to be simulated from A
		@param regions: The boundaries for the regions that each thread is responsible for
		@param context: Information necessary to understand the given inputs, containing: [y_dim, x_dim, numSegments]
		@param rules: The ruleset to be used to step forward through the given simulation
	 */
	template <typename T>
	__global__ void stepForwardRegion(T* A, T* B, int* regions, int* context, IRulesArray<T>* rules) {
		// context: y_dim, x_dim, numSegments
		const int tid = threadIdx.x + blockDim.x * blockIdx.x;

		if(tid >= NUM_SEGMENTS)
		{
			// if there isn't the data for the thread to read, end
			return;
		}
		for (int y = regions[tid * 4]; y <= regions[tid * 4 + 1]; ++y) {
			for (int x = regions[tid * 4 + 2]; x <= regions[tid * 4 + 3]; ++x) {
				if(y == -1 && x == -1)
				{
					return;
				}
				B[x + y * X_DIM] = rules->getNextState(A, y, x);
			}
		}
	}

	template <typename T>
	double SimulatorGPU<T>::stepForward(int steps) {
		this->timer.reset();
		// declare the variables needed
		int numSegments = this->nBlocks * this->nThreads;
		T *h_currFrame, *h_newFrame, *d_currFrame, *d_newFrame;
		int *h_segments, *d_segments;
		int *h_context,*d_context;
		int* h_dimensions, *d_dimensions;
		int frameSize = this->x_dim * this->y_dim;

		// define the host variables
		h_segments = this->segmenter.segmentToArray(this->y_dim, this->x_dim, numSegments);
		h_context = static_cast<int*>(malloc(sizeof(int) * 3));
		h_context[0] = this->y_dim;
		h_context[1] = this->x_dim;
		h_context[2] = numSegments;

		h_currFrame = this->cellStore.back();
		h_dimensions = static_cast<int*>(malloc(sizeof(int) * 2));
		h_dimensions[0] = this->x_dim;
		h_dimensions[1] = this->y_dim;
		// allocate the device memory
		checkCudaErrors(hipMalloc(&d_currFrame, sizeof(T) * frameSize));
		checkCudaErrors(hipMalloc(&d_newFrame, sizeof(T) * frameSize));
		checkCudaErrors(hipMalloc(&d_segments, sizeof(int) * 4 * numSegments));
		checkCudaErrors(hipMalloc(&d_context, sizeof(int) * 3));
		checkCudaErrors(hipMalloc(&d_dimensions, sizeof(int) * 2));
		
		// copy over data to the device
		checkCudaErrors(hipMemcpy(d_currFrame, h_currFrame, sizeof(T) * frameSize, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_context, h_context, sizeof(int) * 3, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_segments, h_segments, sizeof(int) * 4 * numSegments, hipMemcpyHostToDevice));
		hipMemcpy(d_dimensions, h_dimensions, sizeof(int) * 2, hipMemcpyHostToDevice);
		// Get the ruleset 
		RulesArrayConway<T>* h_con = dynamic_cast<RulesArrayConway<T>*>(&(this->rules));
		RulesArrayBML<T>* h_bml = dynamic_cast<RulesArrayBML<T>*>(&(this->rules));

		if (h_con != nullptr)
		{
			// Rules type is Conway
			// Get the rules set up on the device
			RulesArrayConway<T>* d_con;
			checkCudaErrors(hipMalloc(&d_con, sizeof(RulesArrayConway<T>) * 2));
			constructConway<T><<<1,1>>>(d_con, d_dimensions);
			
			for (int step = 0; step < steps; ++step)
			{
				this->blankFrame();
				h_newFrame = this->cellStore.back();
				//no need to copy the new frame to the device, as every cell's value will be assigned to during the step process
				stepForwardRegion<int> << <nBlocks, nThreads >> > (d_currFrame, d_newFrame, d_segments, d_context, d_con);
				// copy back the data 
				hipMemcpy(h_newFrame, d_newFrame, sizeof(T) * frameSize, hipMemcpyDeviceToHost);

				// swap the pointers, ready for the next iteration
				T *temp = d_currFrame;
				d_currFrame = d_newFrame;
				d_newFrame = temp;
			}
			// Free up the space used by the ruleset
			checkCudaErrors(hipFree(d_con));
		}
		else if((h_bml != nullptr))
		{
			// Rules type is Conway
			// Get the rules set up on the device
			RulesArrayBML<T>* d_bml;
			checkCudaErrors(hipMalloc(&d_bml, sizeof(RulesArrayBML<T>) * 2));
			constructBML<T> << <1, 1 >> > (d_bml, d_dimensions);

			for (int step = 0; step < steps; ++step)
			{
				this->blankFrame();
				h_newFrame = this->cellStore.back();
				//no need to copy the new frame to the device, as every cell's value will be assigned to during the step process
				stepForwardRegion<int> << <nBlocks, nThreads >> > (d_currFrame, d_newFrame, d_segments, d_context, d_bml);
				// copy back the data 
				hipMemcpy(h_newFrame, d_newFrame, sizeof(T) * frameSize, hipMemcpyDeviceToHost);

				// swap the pointers, ready for the next iteration
				T *temp = d_currFrame;
				d_currFrame = d_newFrame;
				d_newFrame = temp;
			}
			// Free up the space used by the ruleset
			checkCudaErrors(hipFree(d_bml));
		}

		// Free up all the space used by the function call
		checkCudaErrors(hipFree(d_currFrame));
		checkCudaErrors(hipFree(d_newFrame));
		checkCudaErrors(hipFree(d_context));
		checkCudaErrors(hipFree(d_segments));
		checkCudaErrors(hipFree(d_dimensions));

		//checkCudaErrors(hipDeviceReset());
		double elapsed = this->timer.elapsed();
		this->elapsedTime += elapsed;
		return elapsed;
	}

	template class SimulatorGPU<int>;
}
