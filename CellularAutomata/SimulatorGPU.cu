#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "SimulatorGPU.hpp"
#include "IRulesArray.hpp"
#include <>
#include "hip/hip_runtime_api.h"
#include <crt/host_defines.h>

namespace CellularAutomata {

	template <typename T>
	__global__ void stepForwardRegion(T* A, T* B, int* regions, const IRulesArray<T>* rules) {
		int y_dim = A[0];
		int x_dim = A[1];
		int tid = threadIdx.x + blockDim.x * blockIdx.x;
		for (int y = regions[tid * 4]; y < regions[tid * 4 + 1]; ++y) {
			for (int x = regions[tid * 4 + 2]; x < regions[tid * 4 + 3]; ++x) {
				B[x + y * x_dim] = rules->getNextState(A, y, x);
			}
		}
	}

	template <typename T>
	double SimulatorGPU<T>::stepForward(int steps) {
		this->timer.reset();
		T* currFrame = cellStore.back();
		this->blankFrame();
		T* newFrame = cellStore.back();
		int frameSize = x_dim * y_dim;
		checkCudaErrors(hipMallocManaged(&currFrame, sizeof(T) * frameSize));
		checkCudaErrors(hipMallocManaged(&newFrame, sizeof(T) * frameSize));
		int* segments = this->segmenter.segmentToArray(this->y_dim, this->x_dim, this->nBlocks * this->nThreads);
		checkCudaErrors(hipMallocManaged(&segments, sizeof(int) * 4 * this->nThreads * this->nBlocks));
		const IRulesArray<T>* rules = &this->rules;
		checkCudaErrors(hipMallocManaged(&rules, sizeof(rules)));
		//stepForwardRegion<int> <<<nBlocks, nThreads>>> (currFrame, newFrame, segments, rules);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipFree(newFrame));
		checkCudaErrors(hipFree(currFrame));
		checkCudaErrors(hipFree(segments));
		hipDeviceReset();
		double elapsed = this->timer.elapsed();
		this->elapsedTime += elapsed;
		return elapsed;
	}

	template class SimulatorGPU<int>;
}
